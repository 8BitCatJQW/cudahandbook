#include "hip/hip_runtime.h"
/*
 *
 * nbody.cu
 *
 * N-body example that illustrates gravitational simulation.
 * This is the type of computation that GPUs excel at:
 * parallelizable, with lots of FLOPS per unit of external 
 * memory bandwidth required.
 *
 * Build with: nvcc -I ../chLib <options> nbody.cu
 * Requires: No minimum SM requirement.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <stdio.h>
#include <conio.h>

#include <math.h>

#include <chCommandLine.h>
#include <chError.h>
#include <chTimer.h>

#include "bodybodyInteraction.cuh"
#include "bodybodyInteraction_SSE.h"

inline void
randomVector( float v[3] )
{
    float lenSqr;
    do {
        v[0] = rand() / (float) RAND_MAX * 2 - 1;
        v[1] = rand() / (float) RAND_MAX * 2 - 1;
        v[2] = rand() / (float) RAND_MAX * 2 - 1;
        lenSqr = v[0]*v[0]+v[1]*v[1]+v[2]*v[2];
    } while ( lenSqr > 1.0f );
}

void
randomUnitBodies( float *pos, float *vel, size_t N )
{
    for ( size_t i = 0; i < N; i++ ) {
        randomVector( &pos[4*i] );
        randomVector( &vel[4*i] );
        pos[4*i+3] = 1.0f;  // unit mass
        vel[4*i+3] = 1.0f;
    }
}

template<typename T>
static float
relError( float a, float b )
{
    if ( a == b ) return 0.0f;
    return fabsf(a-b)/b;
}

float *g_hostAOS_PosMass;
float *g_hostAOS_VelInvMass;
float *g_hostAOS_Force;

float *g_dptrAOS_PosMass;
float *g_dptrAOS_Force;


// Buffer to hold the golden version of the forces, used for comparison
// Along with timing results, we report the maximum relative error with 
// respect to this array.
float *g_hostAOS_Force_Golden;

float *g_hostSOA_Pos[3];
float *g_hostSOA_Force[3];
float *g_hostSOA_Mass;
float *g_hostSOA_InvMass;

size_t g_N;

float g_softening = 0.1f;
float g_damping = 0.995f;
float g_dt = 0.016f;

template<typename T>
static T
relError( T a, T b )
{
    if ( a == b ) return 0.0f;
    T relErr = (a-b)/b;
    // Manually take absolute value
    return (relErr<0.0f) ? -relErr : relErr;
}

#include "nbody_CPU_AOS.h"
#include "nbody_CPU_SOA.h"
#include "nbody_CPU_SSE.h"
#include "nbody_GPU_AOS.cuh"
#include "nbody_GPU_Shared.cuh"
#include "nbody_GPU_Shuffle.cuh"
#include "nbody_GPU_Atomic.cuh"

void
integrateGravitation_AOS( float *ppos, float *pvel, float *pforce, float dt, float damping, size_t N )
{
    for ( size_t i = 0; i < N; i++ ) {
        int index = 4*i;
        int indexForce = 3*i;

        float pos[3], vel[3], force[3];
        pos[0] = ppos[index+0];
        pos[1] = ppos[index+1];
        pos[2] = ppos[index+2];
        float invMass = pvel[index+3];

        vel[0] = pvel[index+0];
        vel[1] = pvel[index+1];
        vel[2] = pvel[index+2];

        force[0] = pforce[indexForce+0];
        force[1] = pforce[indexForce+1];
        force[2] = pforce[indexForce+2];

        // acceleration = force / mass;
        // new velocity = old velocity + acceleration * deltaTime
        vel[0] += (force[0] * invMass) * dt;
        vel[1] += (force[1] * invMass) * dt;
        vel[2] += (force[2] * invMass) * dt;

        vel[0] *= damping;
        vel[1] *= damping;
        vel[2] *= damping;

        // new position = old position + velocity * deltaTime
        pos[0] += vel[0] * dt;
        pos[1] += vel[1] * dt;
        pos[2] += vel[2] * dt;

        ppos[index+0] = pos[0];
        ppos[index+1] = pos[1];
        ppos[index+2] = pos[2];

        pvel[index+0] = vel[0];
        pvel[index+1] = vel[1];
        pvel[index+2] = vel[2];
    }
}

enum nbodyAlgorithm_enum {
    CPU_AOS = 0,    /* This is the golden implementation */
    CPU_SOA,
    CPU_SSE,
    GPU_AOS,
    GPU_Atomic,
    GPU_Shared,
    GPU_Shuffle/*,
    GPU_SOA,
    MultiGPU*/
};

const char *rgszAlgorithmNames[] = { "CPU_AOS", "CPU_SOA", "CPU_SSE", "GPU_AOS", "GPU_Atomic", "GPU_Shared", "GPU_Shuffle" };

enum nbodyAlgorithm_enum g_Algorithm = CPU_SSE;
bool g_bCrossCheck = true;

bool
ComputeGravitation( 
    float *ms,
    float *maxRelError,
    nbodyAlgorithm_enum algorithm, 
    bool bCrossCheck )
{
    hipError_t status;
    bool bSOA = false;

    if ( bCrossCheck ) {
        ComputeGravitation_AOS( 
            g_hostAOS_Force_Golden,
            g_hostAOS_PosMass,
            g_softening*g_softening,
            g_N );
    }

    // AOS -> SOA data structures in case we are measuring SOA performance
    for ( size_t i = 0; i < g_N; i++ ) {
        g_hostSOA_Pos[0][i]  = g_hostAOS_PosMass[4*i+0];
        g_hostSOA_Pos[1][i]  = g_hostAOS_PosMass[4*i+1];
        g_hostSOA_Pos[2][i]  = g_hostAOS_PosMass[4*i+2];
        g_hostSOA_Mass[i]    = g_hostAOS_PosMass[4*i+3];
        g_hostSOA_InvMass[i] = 1.0f / g_hostSOA_Mass[i];
    }

    // CPU->GPU copies in case we are measuring GPU performance
    CUDART_CHECK( hipMemcpyAsync( g_dptrAOS_PosMass, g_hostAOS_PosMass, 4*g_N*sizeof(float), hipMemcpyHostToDevice ) );

    switch ( algorithm ) {
        case CPU_AOS:
            *ms = ComputeGravitation_AOS( 
                g_hostAOS_Force,
                g_hostAOS_PosMass,
                g_softening*g_softening,
                g_N );
            break;
        case CPU_SOA:
            *ms = ComputeGravitation_SOA(
                g_hostSOA_Force,
                g_hostSOA_Pos,
                g_hostSOA_Mass,
                g_softening*g_softening,
                g_N );
            bSOA = true;
            break;
        case CPU_SSE:
            *ms = ComputeGravitation_SSE(
                g_hostSOA_Force,
                g_hostSOA_Pos,
                g_hostSOA_Mass,
                g_softening*g_softening,
                g_N );
            bSOA = true;
            break;
        case GPU_AOS:
            *ms = ComputeGravitation_GPU_AOS( 
                g_dptrAOS_Force,
                g_dptrAOS_PosMass,
                g_softening*g_softening,
                g_N );
            CUDART_CHECK( hipMemcpy( g_hostAOS_Force, g_dptrAOS_Force, 3*g_N*sizeof(float), hipMemcpyDeviceToHost ) );
            break;
        case GPU_Atomic:
            hipMemset( g_dptrAOS_Force, 0, 3*sizeof(float) );
            *ms = ComputeGravitation_GPU_Atomic( 
                g_dptrAOS_Force,
                g_dptrAOS_PosMass,
                g_softening*g_softening,
                g_N );
            CUDART_CHECK( hipMemcpy( g_hostAOS_Force, g_dptrAOS_Force, 3*g_N*sizeof(float), hipMemcpyDeviceToHost ) );
            break;
        case GPU_Shared:
            *ms = ComputeGravitation_GPU_Shared( 
                g_dptrAOS_Force,
                g_dptrAOS_PosMass,
                g_softening*g_softening,
                g_N );
            CUDART_CHECK( hipMemcpy( g_hostAOS_Force, g_dptrAOS_Force, 3*g_N*sizeof(float), hipMemcpyDeviceToHost ) );
            break;
        case GPU_Shuffle:
            *ms = ComputeGravitation_GPU_Shuffle( 
                g_dptrAOS_Force,
                g_dptrAOS_PosMass,
                g_softening*g_softening,
                g_N );
            CUDART_CHECK( hipMemcpy( g_hostAOS_Force, g_dptrAOS_Force, 3*g_N*sizeof(float), hipMemcpyDeviceToHost ) );
            break;
    }

    // SOA -> AOS
    if ( bSOA ) {
        for ( size_t i = 0; i < g_N; i++ ) {
            g_hostAOS_Force[3*i+0] = g_hostSOA_Force[0][i];
            g_hostAOS_Force[3*i+1] = g_hostSOA_Force[1][i];
            g_hostAOS_Force[3*i+2] = g_hostSOA_Force[2][i];
        }
    }

    if ( bCrossCheck ) {
        float max = 0.0f;
        for ( size_t i = 0; i < 3*g_N; i++ ) {
            float err = relError( g_hostAOS_Force[i], g_hostAOS_Force_Golden[i] );
            if ( err > max ) {
                max = err;
            }
        }
        *maxRelError = max;
    }

    integrateGravitation_AOS( 
        g_hostAOS_PosMass,
        g_hostAOS_VelInvMass,
        g_hostAOS_Force,
        g_dt,
        g_damping,
        g_N );
    return true;
Error:
    return false;
}

int
main( int argc, char *argv[] )
{
    hipError_t status;
    // kiloparticles
    int kParticles = 4;

    CUDART_CHECK( hipSetDeviceFlags( hipDeviceMapHost ) );

    chCommandLineGet( &kParticles, "numbodies", argc, argv );
    g_N = kParticles*1024;
    printf( "Running simulation with %d particles\n", (int) g_N );

    CUDART_CHECK( hipHostAlloc( (void **) &g_hostAOS_PosMass, 4*g_N*sizeof(float), hipHostMallocPortable ) );
    for ( int i = 0; i < 3; i++ ) {
        CUDART_CHECK( hipHostAlloc( (void **) &g_hostSOA_Pos[i], g_N*sizeof(float), hipHostMallocPortable ) );
        CUDART_CHECK( hipHostAlloc( (void **) &g_hostSOA_Force[i], g_N*sizeof(float), hipHostMallocPortable ) );
    }
    CUDART_CHECK( hipHostAlloc( (void **) &g_hostAOS_Force, 3*g_N*sizeof(float), hipHostMallocPortable ) );
    CUDART_CHECK( hipHostAlloc( (void **) &g_hostAOS_Force_Golden, 3*g_N*sizeof(float), hipHostMallocPortable ) );
    CUDART_CHECK( hipHostAlloc( (void **) &g_hostAOS_VelInvMass, 4*g_N*sizeof(float), hipHostMallocPortable ) );
    CUDART_CHECK( hipHostAlloc( (void **) &g_hostSOA_Mass, g_N*sizeof(float), hipHostMallocPortable ) );
    CUDART_CHECK( hipHostAlloc( (void **) &g_hostSOA_InvMass, g_N*sizeof(float), hipHostMallocPortable ) );

    CUDART_CHECK( hipMalloc( &g_dptrAOS_PosMass, 4*g_N*sizeof(float) ) );
    CUDART_CHECK( hipMalloc( (void **) &g_dptrAOS_Force, 3*g_N*sizeof(float) ) );

    randomUnitBodies( g_hostAOS_PosMass, g_hostAOS_VelInvMass, g_N );
    for ( size_t i = 0; i < g_N; i++ ) {
        g_hostSOA_Mass[i] = g_hostAOS_PosMass[4*i+3];
        g_hostSOA_InvMass[i] = 1.0f / g_hostSOA_Mass[i];
    }

    while ( ! kbhit() ) {
        float ms, err;
        ComputeGravitation( &ms, &err, g_Algorithm, g_bCrossCheck );
        double interactionsPerSecond = (double) g_N*g_N*1000.0f / ms;
        if ( interactionsPerSecond > 1e9 ) {
            printf ( "%s: %.2f ms = %.3fx10^9 interactions/s (Rel. error: %E)\n", 
                rgszAlgorithmNames[g_Algorithm], 
                ms, 
                interactionsPerSecond/1e9, 
                err );
        }
        else {
            printf ( "%s: %.2f ms = %.3fx10^6 interactions/s (Rel. error: %E)\n", 
                rgszAlgorithmNames[g_Algorithm], 
                ms, 
                interactionsPerSecond/1e6, 
                err );
        }
    }

    return 0;
Error:
    if ( hipSuccess != status ) {
        printf( "CUDA Error: %s\n", hipGetErrorString( status ) );
    }
    return 1;
}
