#include "hip/hip_runtime.h"
/*
 *
 * nbody.cu
 *
 * N-body example that illustrates gravitational simulation.
 * This is the type of computation that GPUs excel at:
 * parallelizable, with lots of FLOPS per unit of external 
 * memory bandwidth required.
 *
 * Build with: nvcc -I ../chLib <options> nbody.cu
 * Requires: No minimum SM requirement.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <stdio.h>
#include <conio.h>

#include <math.h>

#include <chCommandLine.h>
#include <chError.h>
#include <chTimer.h>

inline void
randomVector( float v[3] )
{
    float lenSqr;
    do {
        v[0] = rand() / (float) RAND_MAX * 2 - 1;
        v[1] = rand() / (float) RAND_MAX * 2 - 1;
        v[2] = rand() / (float) RAND_MAX * 2 - 1;
        lenSqr = v[0]*v[0]+v[1]*v[1]+v[2]*v[2];
    } while ( lenSqr > 1.0f );
}

void
randomUnitBodies( float *pos, float *vel, size_t N )
{
    for ( size_t i = 0; i < N; i++ ) {
        randomVector( &pos[4*i] );
        randomVector( &vel[4*i] );
        pos[4*i+3] = 1.0f;  // unit mass
        vel[4*i+3] = 1.0f;
    }
}

float *g_hostAOS_PosMass[2];
float *g_hostAOS_VelInvMass;
float *g_hostAOS_Force;

float *g_hostSOA_PosX[2];
float *g_hostSOA_PosY[2];
float *g_hostSOA_PosZ[2];
float *g_hostSOA_Mass;
float *g_hostSOA_InvMass;

size_t g_N;


float g_softening = 0.1f;
float g_damping = 0.995f;
float g_dt = 0.016f;


template <typename T>
__host__ __device__ void bodyBodyInteraction(
    T accel[3], 
    T x0, T y0, T z0,
    T x1, T y1, T z1, T mass1, 
    T softeningSquared)
{
    T dx = x1 - x0;
    T dy = y1 - y0;
    T dz = z1 - z0;

    T distSqr = dx*dx + dy*dy + dz*dz;
    distSqr += softeningSquared;

    T invDist = (T)1.0 / (T)sqrt((double)distSqr);

    T invDistCube =  invDist * invDist * invDist;
    T s = mass1 * invDistCube;

    accel[0] += dx * s;
    accel[1] += dy * s;
    accel[2] += dz * s;
}

template<typename T>
static T
relError( T a, T b )
{
    if ( a == b ) return 0.0f;
    T relErr = (a-b)/b;
    // Manually take absolute value
    return (relErr<0.0f) ? -relErr : relErr;
}



float
ComputeGravitation_AOS( 
    float *force, 
    float *posMass,
    float softeningSquared,
    size_t N
)
{
    chTimerTimestamp start, end;
    chTimerGetTime( &start );
    for (size_t i = 0; i < N; i++)
    {
        float acc[3] = {0, 0, 0};
        float myX = posMass[i*4+0];
        float myY = posMass[i*4+1];
        float myZ = posMass[i*4+2];

        for ( size_t j = 0; j < N; j++ ) {
            float bodyX = posMass[j*4+0];
            float bodyY = posMass[j*4+1];
            float bodyZ = posMass[j*4+2];
            float bodyMass = posMass[j*4+3];

            bodyBodyInteraction<float>(
                acc, 
                myX, myY, myZ,
                bodyX, bodyY, bodyZ, bodyMass,
                softeningSquared );
        }

        force[3*i+0] = acc[0];
        force[3*i+1] = acc[1];
        force[3*i+2] = acc[2];
    }
    chTimerGetTime( &end );
    return (float) chTimerElapsedTime( &start, &end ) * 1000.0f;
}

void
integrateGravitation_AOS( float *ppos, float *pvel, float *pforce, float dt, float damping, size_t N )
{
    for ( size_t i = 0; i < N; i++ ) {
        int index = 4*i;
        int indexForce = 3*i;

        float pos[3], vel[3], force[3];
        pos[0] = ppos[index+0];
        pos[1] = ppos[index+1];
        pos[2] = ppos[index+2];
        float invMass = pvel[index+3];

        vel[0] = pvel[index+0];
        vel[1] = pvel[index+1];
        vel[2] = pvel[index+2];

        force[0] = pforce[indexForce+0];
        force[1] = pforce[indexForce+1];
        force[2] = pforce[indexForce+2];

        // acceleration = force / mass;
        // new velocity = old velocity + acceleration * deltaTime
        vel[0] += (force[0] * invMass) * dt;
        vel[1] += (force[1] * invMass) * dt;
        vel[2] += (force[2] * invMass) * dt;

        vel[0] *= damping;
        vel[1] *= damping;
        vel[2] *= damping;

        // new position = old position + velocity * deltaTime
        pos[0] += vel[0] * dt;
        pos[1] += vel[1] * dt;
        pos[2] += vel[2] * dt;

        ppos[index+0] = pos[0];
        ppos[index+1] = pos[1];
        ppos[index+2] = pos[2];

        pvel[index+0] = vel[0];
        pvel[index+1] = vel[1];
        pvel[index+2] = vel[2];
    }
}


int
main( int argc, char *argv[] )
{
    hipError_t status;
    // kiloparticles
    int kParticles = 4;

    CUDART_CHECK( hipSetDeviceFlags( hipDeviceMapHost ) );

    chCommandLineGet( &kParticles, "numbodies", argc, argv );
    g_N = kParticles*1024;
    printf( "Running simulation with %d particles\n", (int) g_N );

    for ( int i = 0; i < 2; i++ ) {
        CUDART_CHECK( hipHostAlloc( (void **) &g_hostAOS_PosMass[i], 4*g_N*sizeof(float), hipHostMallocPortable ) );
        CUDART_CHECK( hipHostAlloc( (void **) &g_hostSOA_PosX[i], g_N*sizeof(float), hipHostMallocPortable ) );
        CUDART_CHECK( hipHostAlloc( (void **) &g_hostSOA_PosY[i], g_N*sizeof(float), hipHostMallocPortable ) );
        CUDART_CHECK( hipHostAlloc( (void **) &g_hostSOA_PosZ[i], g_N*sizeof(float), hipHostMallocPortable ) );
    }
    CUDART_CHECK( hipHostAlloc( (void **) &g_hostAOS_Force, 3*g_N*sizeof(float), hipHostMallocPortable ) );
    CUDART_CHECK( hipHostAlloc( (void **) &g_hostAOS_VelInvMass, 4*g_N*sizeof(float), hipHostMallocPortable ) );
    CUDART_CHECK( hipHostAlloc( (void **) &g_hostSOA_Mass, g_N*sizeof(float), hipHostMallocPortable ) );
    CUDART_CHECK( hipHostAlloc( (void **) &g_hostSOA_InvMass, g_N*sizeof(float), hipHostMallocPortable ) );
    randomUnitBodies( g_hostAOS_PosMass[0], g_hostAOS_VelInvMass, g_N );

    while ( ! kbhit() ) {
        float ms = ComputeGravitation_AOS( 
            g_hostAOS_Force,
            g_hostAOS_PosMass[0],
            g_softening*g_softening,
            g_N );
        integrateGravitation_AOS( 
            g_hostAOS_PosMass[0],
            g_hostAOS_VelInvMass,
            g_hostAOS_Force,
            g_dt,
            g_damping,
            g_N );
        double interactionsPerSecond = (double) g_N*g_N*1000.0f / ms;
        printf ( "%.2f ms = %.2f Minteractions/s\n", ms, interactionsPerSecond/1e6 );
    }

    return 0;
Error:
    if ( hipSuccess != status ) {
        printf( "CUDA Error: %s\n", hipGetErrorString( status ) );
    }
    return 1;
}
