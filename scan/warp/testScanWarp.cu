#include "hip/hip_runtime.h"
/*
 *
 * testScanWarp.cu
 *
 * Microdemo to test warp scan algorithms.
 *
 * Build with: nvcc -I ..\chLib <options> testScanWarp.cu
 * Requires: No minimum SM requirement.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met:
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <stdlib.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <chAssert.h>
#include <chError.h>

typedef unsigned int uint;

#include "scanWarp.cuh"
#include "scanWarp2.cuh"
#include "scanWarpShuffle.cuh"

#define min(a,b) ((a)<(b)?(a):(b))

int *g_hostIn, *g_hostOut;


enum ScanType {
    Inclusive, Exclusive
};

template<int period>
void
ScanExclusiveCPUPeriodic( int *out, const int *in, size_t N )
{
    for ( size_t i = 0; i < N; i += period ) {
        int sum = 0;
        for ( size_t j = 0; j < period; j++ ) {
            int next = in[i+j]; // in case we are doing this in place
            out[i+j] = sum;
            sum += next;
        }
    }
}

template<int period>
void
ScanInclusiveCPUPeriodic( int *out, const int *in, size_t N )
{
    for ( size_t i = 0; i < N; i += period ) {
        int sum = 0;
        for ( size_t j = 0; j < period; j++ ) {
            sum += in[i+j];
            out[i+j] = sum;
        }
    }
}

void
RandomArray( int *out, size_t N, int modulus )
{
    for ( size_t i = 0; i < N; i++ ) {
        out[i] = rand() % modulus;
    }
}

__global__ void
ScanInclusiveGPUWarp( int *out, const int *in, size_t N )
{
    extern __shared__ int sPartials[];
    for ( size_t i = blockIdx.x*blockDim.x;
                 i < N;
                 i += blockDim.x ) {
        sPartials[threadIdx.x] = in[i+threadIdx.x];
        __syncthreads();
        out[i+threadIdx.x] = scanWarp<int,false>( sPartials+threadIdx.x );
    }
}

void
ScanInclusiveGPU( 
    int *out, 
    const int *in, 
    size_t N, 
    int cThreads )
{
    int cBlocks = (int) (N/150);
    if ( cBlocks > 150 ) {
        cBlocks = 150;
    }
    ScanInclusiveGPUWarp<<<cBlocks, cThreads, cThreads*sizeof(int)>>>( 
        out, in, N );
}

__global__ void
ScanInclusiveGPUWarp2( int *out, const int *in, size_t N )
{
    extern __shared__ int sPartials[];
    for ( size_t i = blockIdx.x*blockDim.x;
                 i < N;
                 i += blockDim.x ) {
        sPartials[threadIdx.x] = in[i+threadIdx.x];
        __syncthreads();
        out[i+threadIdx.x] = scanWarp2<int,false>( sPartials+threadIdx.x );
    }
}

void
ScanInclusiveGPU2( 
    int *out, 
    const int *in, 
    size_t N, 
    int cThreads )
{
    int cBlocks = (int) (N/150);
    if ( cBlocks > 150 ) {
        cBlocks = 150;
    }
    ScanInclusiveGPUWarp2<<<cBlocks, cThreads, cThreads*sizeof(int)>>>( 
            out, in, N );
}

__global__ void
ScanInclusiveGPUWarpShuffle( int *out, const int *in, size_t N )
{
    extern __shared__ int sPartials[];
    for ( size_t i = blockIdx.x*blockDim.x;
                 i < N;
                 i += blockDim.x ) {
        out[i+threadIdx.x] = inclusive_scan_warp_shfl<5>( in[i+threadIdx.x] );
    }
}

void
ScanInclusiveGPUShuffle( 
    int *out, 
    const int *in, 
    size_t N, 
    int cThreads )
{
    int cBlocks = (int) (N/150);
    if ( cBlocks > 150 ) {
        cBlocks = 150;
    }
    ScanInclusiveGPUWarpShuffle<<<cBlocks, cThreads>>>( out, in, N );
}

template<class T>
bool
TestScanWarp( 
    float *pMelementspersecond,
    const char *szScanFunction, 
    void (*pfnScanCPU)(T *, const T *, size_t),
    void (*pfnScanGPU)(T *, const T *, size_t, int), 
    size_t N, 
    int numThreads )
{
    bool ret = false;
    hipError_t status;
    int *inGPU = 0;
    int *outGPU = 0;
    int *inCPU = (T *) malloc( N*sizeof(T) );
    int *outCPU = (int *) malloc( N*sizeof(T) );
    int *hostGPU = (int *) malloc( N*sizeof(T) );
    hipEvent_t evStart = 0, evStop = 0;
    if ( 0==inCPU || 0==outCPU || 0==hostGPU )
        goto Error;

    printf( "Testing %s (%d threads/block)\n", szScanFunction, numThreads );

    CUDART_CHECK( hipEventCreate( &evStart ) );
    CUDART_CHECK( hipEventCreate( &evStop ) );
    CUDART_CHECK( hipMalloc( &inGPU, N*sizeof(T) ) );
    CUDART_CHECK( hipMalloc( &outGPU, N*sizeof(T) ) );
    CUDART_CHECK( hipMemset( inGPU, 0, N*sizeof(T) ) );
    CUDART_CHECK( hipMemset( outGPU, 0, N*sizeof(T) ) );

    CUDART_CHECK( hipMemset( outGPU, 0, N*sizeof(T) ) );

    RandomArray( inCPU, N, 256 );
for ( int i = 0; i < N; i++ ) {
    inCPU[i] = i;
}
    
    pfnScanCPU( outCPU, inCPU, N );
g_hostIn = inCPU;

    CUDART_CHECK( hipMemcpy( inGPU, inCPU, N*sizeof(T), hipMemcpyHostToDevice ) );
    CUDART_CHECK( hipEventRecord( evStart, 0 ) );
    pfnScanGPU( outGPU, inGPU, N, numThreads );
    CUDART_CHECK( hipEventRecord( evStop, 0 ) );
    CUDART_CHECK( hipMemcpy( hostGPU, outGPU, N*sizeof(T), hipMemcpyDeviceToHost ) );
    for ( size_t i = 0; i < N; i++ ) {
        if ( hostGPU[i] != outCPU[i] ) {
            printf( "Scan failed\n" );
#ifdef _WIN32
            __debugbreak();//_asm int 3
#else
            assert(0);
#endif
            goto Error;
        }
    }
    {
        float ms;
        CUDART_CHECK( hipEventElapsedTime( &ms, evStart, evStop ) );
        double Melements = N/1e6;
        *pMelementspersecond = 1000.0f*Melements/ms;
    }
    ret = true;
Error:
    hipEventDestroy( evStart );
    hipEventDestroy( evStop );
    hipFree( outGPU );
    hipFree( inGPU );
    free( inCPU );
    free( outCPU );
    free( hostGPU );
    return ret;
}

int
main( int argc, char *argv[] )
{
    hipError_t status;
    int maxThreads;
    int numInts = 32*1048576;

    CUDART_CHECK( hipSetDevice( 0 ) );
    CUDART_CHECK( hipSetDeviceFlags( hipDeviceMapHost ) );

    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties( &prop, 0 );
        maxThreads = prop.maxThreadsPerBlock;
    }

#define SCAN_TEST_VECTOR( CPUFunction, GPUFunction, N, numThreads ) do { \
    float fMelementsPerSecond; \
    srand(0); \
    bool bSuccess = TestScanWarp<int>( &fMelementsPerSecond, #GPUFunction, CPUFunction, GPUFunction, N, numThreads ); \
    if ( ! bSuccess ) { \
        printf( "%s failed: N=%d, numThreads=%d\n", #GPUFunction, N, numThreads ); \
        exit(1); \
    } \
    if ( fMelementsPerSecond > maxElementsPerSecond ) { \
        maxElementsPerSecond = fMelementsPerSecond; \
    } \
\
} while (0)

    printf( "Problem size: %d integers\n", numInts );

    for ( int numThreads = 256; numThreads <= maxThreads; numThreads *= 2 ) {
        float maxElementsPerSecond = 0.0f;
        SCAN_TEST_VECTOR( ScanInclusiveCPUPeriodic<32>, ScanInclusiveGPU, numInts, numThreads );
        printf( "GPU: %.2f Melements/s\n", maxElementsPerSecond );
        maxElementsPerSecond = 0.0f;
        SCAN_TEST_VECTOR( ScanInclusiveCPUPeriodic<32>, ScanInclusiveGPU2, numInts, numThreads );
        printf( "GPU2: %.2f Melements/s\n", maxElementsPerSecond );
        maxElementsPerSecond = 0.0f;
        SCAN_TEST_VECTOR( ScanInclusiveCPUPeriodic<32>, ScanInclusiveGPUShuffle, numInts, numThreads );
        printf( "Shuffle: %.2f Melements/s\n", maxElementsPerSecond );
    }

    return 0;
Error:
    return 1;
}
