#include "hip/hip_runtime.h"
/*
 *
 * peer2peerMemcpy.cu
 *
 * Sample shows how to use portable pinned memory and inter-
 * GPU synchronization to perform a peer-to-peer memcpy.
 *
 * Build with: nvcc -I ../chLib <options> peer2peerMemcpy.cu
 * Requires: No minimum SM requirement.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */


#include <stdio.h>

#include "chError.h"
#include "chTimer.h"

#define MAX_DEVICES 32

#define STAGING_BUFFER_SIZE 1048576

void *g_hostBuffers[MAX_DEVICES];

// Indexed as follows: [device][event]
hipEvent_t g_events[MAX_DEVICES][2];
bool g_bEnabled[MAX_DEVICES][MAX_DEVICES];

// these are already defined on some platforms - make our
// own definitions that will work.
#undef min
#undef max
#define min(a,b) ((a)<(b)?(a):(b))
#define max(a,b) ((b)<(a)?(a):(b))

hipError_t
chMemcpyPeerToPeer( 
    void *_dst, int dstDevice, 
    const void *_src, int srcDevice, 
    size_t N ) 
{
    hipError_t status;
    char *dst = (char *) _dst;
    const char *src = (const char *) _src;
    int stagingIndex = 0;
    while ( N ) {
        size_t thisCopySize = min( N, STAGING_BUFFER_SIZE );

        CUDART_CHECK( hipSetDevice( srcDevice ) );
        CUDART_CHECK( hipStreamWaitEvent( NULL, g_events[dstDevice][stagingIndex], 0 ) );
        CUDART_CHECK( hipMemcpyAsync( g_hostBuffers[stagingIndex], src, thisCopySize, 
            hipMemcpyDeviceToHost, NULL ) );
        CUDART_CHECK( hipEventRecord( g_events[srcDevice][stagingIndex] ) );

        CUDART_CHECK( hipSetDevice( dstDevice ) );
        CUDART_CHECK( hipStreamWaitEvent( NULL, g_events[srcDevice][stagingIndex], 0 ) );
        CUDART_CHECK( hipMemcpyAsync( dst, g_hostBuffers[stagingIndex], thisCopySize, 
            hipMemcpyHostToDevice, NULL ) );
        CUDART_CHECK( hipEventRecord( g_events[dstDevice][stagingIndex] ) );

        dst += thisCopySize;
        src += thisCopySize;
        N -= thisCopySize;
        stagingIndex = 1 - stagingIndex;
    }
    // Wait until both devices are done
    CUDART_CHECK( hipSetDevice( srcDevice ) );
    CUDART_CHECK( hipDeviceSynchronize() );

    CUDART_CHECK( hipSetDevice( dstDevice ) );
    CUDART_CHECK( hipDeviceSynchronize() );
    
Error:
    return status;
}

bool
TestMemcpy( 
    int *dst, int dstDevice,
    int *src, int srcDevice,
    int *srcHost, const int *srcOriginal,
    size_t dstOffset, size_t srcOffset, 
    size_t numInts )
{
    hipError_t status;

    memset( srcHost, 0, numInts );
    hipSetDevice( srcDevice );
    CUDART_CHECK( hipMemcpy( src+srcOffset, srcOriginal+srcOffset, 
        numInts*sizeof(int), hipMemcpyHostToDevice ) );
    memset( srcHost, 0, numInts*sizeof(int) );
    chMemcpyPeerToPeer( dst+dstOffset, dstDevice, 
                        src+srcOffset, srcDevice, 
                        numInts*sizeof(int) );
    CUDART_CHECK( hipMemcpy( srcHost, dst+dstOffset, numInts*sizeof(int), hipMemcpyDeviceToHost ) );
    for ( size_t i = 0; i < numInts; i++ ) {
        if ( srcHost[i] != srcOriginal[srcOffset+i] ) {
            return false;
        }
    }
    return true;
Error:
    return false;
}

int
main( int argc, char *argv[] )
{
    int deviceCount;

    hipError_t status;
    int *deviceInt[MAX_DEVICES];
    int *hostInt = 0;
    const size_t numInts = 8*1048576;
    const int cIterations = 10;
    int *testVector = 0;
    printf( "Peer-to-peer memcpy... " ); fflush( stdout );

    chTimerTimestamp start, stop;

    memset( deviceInt, 0, sizeof(deviceInt) );

    CUDART_CHECK( hipGetDeviceCount( &deviceCount ) );

    if ( deviceCount <= 1 ) {
        printf( "Peer-to-peer demo requires at least 2 devices\n" );
        exit(1);
    }

    printf( "%d devices detected\n", deviceCount );

    for ( int i = 0; i < deviceCount; i++ ) {
        hipSetDevice( i );

        CUDART_CHECK( hipEventCreate( &g_events[i][0] ) );
        CUDART_CHECK( hipEventRecord( g_events[i][0], 0 ) );  // so it is signaled on first synchronize
        CUDART_CHECK( hipEventCreate( &g_events[i][1] ) );
        CUDART_CHECK( hipEventRecord( g_events[i][1], 0 ) );  // so it is signaled on first synchronize

        CUDART_CHECK( hipMalloc( &deviceInt[i], numInts*sizeof(int) ) );
    }
    for ( int i = 0; i < deviceCount; i++ ) {
        hipSetDevice( i );
        for ( int j = 0; j < deviceCount; j++ ) {
            if ( i != j ) {
                int bEnabled;
                CUDART_CHECK( hipDeviceCanAccessPeer( &bEnabled, i, j ) );
                g_bEnabled[i][j] = (0 != bEnabled);
                if ( bEnabled ) {
                    CUDART_CHECK( hipDeviceEnablePeerAccess( j, 0 ) );
                }
            }
        }
    }

    CUDART_CHECK( hipHostAlloc( &g_hostBuffers[0], STAGING_BUFFER_SIZE, hipHostMallocPortable ) );
    CUDART_CHECK( hipHostAlloc( &g_hostBuffers[1], STAGING_BUFFER_SIZE, hipHostMallocPortable ) );

    CUDART_CHECK( hipHostAlloc( &hostInt, numInts*sizeof(int), 0 ) );

    testVector = (int *) malloc( numInts*sizeof(int) );
    if ( ! testVector ) {
        printf( "malloc() failed\n" );
        return 1;
    }
    for ( size_t i = 0; i < numInts; i++ ) {
        testVector[i] = rand();
    }

    if ( ! TestMemcpy( deviceInt[0], 0, deviceInt[1], 1, 
                       hostInt, testVector, 0, 0, numInts ) ) {
        goto Error;
    }
    for ( int i = 0; i < cIterations; i++ ) {
        size_t dstOffset = rand() % (numInts-1);
        size_t srcOffset = rand() % (numInts-1);
        size_t intsThisIteration = 1 + rand() % (numInts-max(dstOffset,srcOffset)-1);
        if ( ! TestMemcpy( deviceInt[0], 0, deviceInt[1], 1, hostInt, testVector, dstOffset, srcOffset, intsThisIteration ) ) {
            //TestMemcpy( deviceInt, hostInt, testVector, dstOffset, srcOffset, intsThisIteration );
            goto Error;
        }
    }

    for ( int srcDevice = 0; srcDevice < deviceCount; srcDevice++ ) {
        for ( int dstDevice = 0; dstDevice < deviceCount; dstDevice++ ) {
            if ( srcDevice == dstDevice ) continue;
            printf( "%d <- %d...", dstDevice, srcDevice );
            if ( ! g_bEnabled[srcDevice][dstDevice] ) {
                printf( "Not enabled\n" );
                continue;
            }
            chTimerGetTime( &start );
            for ( int i = 0; i < cIterations; i++ ) {
                hipMemcpyPeerAsync( deviceInt[dstDevice], dstDevice, deviceInt[srcDevice], srcDevice, numInts*sizeof(int) ) ;
            }
            CUDART_CHECK( hipDeviceSynchronize() );
            chTimerGetTime( &stop );

            {
                double MBytes = cIterations*numInts*sizeof(int) / 1048576.0;
                double MBpers = MBytes / chTimerElapsedTime( &start, &stop );

                printf( "%.2f MB/s\n", MBpers );
            }
        }
    }

    hipFree( deviceInt );
    hipHostFree( hostInt );
    return 0;
Error:
    printf( "Error\n" );
    return 1;
}
