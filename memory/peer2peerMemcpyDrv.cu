#include "hip/hip_runtime.h"
/*
 *
 * peer2peerMemcpyDrv.cu
 *
 * Driver API version of the sample shows how to use portable 
 * pinned memory and inter-GPU synchronization to perform a 
 * peer-to-peer memcpy.
 *
 * Build with: nvcc -I ../chLib <options> peer2peerMemcpyDrv.cu
 * Requires: No minimum SM requirement.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */


#include <stdio.h>

#include <hip/hip_runtime.h>

#include "chError.h"
#include "chTimer.h"

#define STAGING_BUFFER_SIZE 1048576

void *g_hostBuffers[2];

// Indexed as follows: [device][event]
hipEvent_t g_events[2][2];

// these are already defined on some platforms - make our
// own definitions that will work.
#undef min
#undef max
#define min(a,b) ((a)<(b)?(a):(b))
#define max(a,b) ((b)<(a)?(a):(b))

hipError_t
chMemcpyPeerToPeer( 
    void *_dst, hipCtx_t dstContext, int dstDevice,
    const void *_src, hipCtx_t srcContext, int srcDevice,
    size_t N ) 
{
    hipError_t status;
    hipDeviceptr_t dst = (hipDeviceptr_t) (intptr_t) _dst;
    hipDeviceptr_t src = (hipDeviceptr_t) (intptr_t) _src;
    int stagingIndex = 0;

    while ( N ) {
        size_t thisCopySize = min( N, STAGING_BUFFER_SIZE );

        CUDA_CHECK( hipCtxPushCurrent( srcContext ) );
        CUDA_CHECK( hipStreamWaitEvent( 
            NULL, g_events[dstDevice][stagingIndex], 0 ) );
        CUDA_CHECK( hipMemcpyDtoHAsync( 
            g_hostBuffers[stagingIndex], 
            src, 
            thisCopySize, 
            NULL ) );
        CUDA_CHECK( hipEventRecord( 
            g_events[srcDevice][stagingIndex], 
            0 ) );

        CUDA_CHECK( hipCtxPopCurrent( &srcContext ) );
        CUDA_CHECK( hipCtxPushCurrent( dstContext ) );
        CUDA_CHECK( hipStreamWaitEvent( 
            NULL, 
            g_events[srcDevice][stagingIndex], 
            0 ) );
        CUDA_CHECK( hipMemcpyHtoDAsync( 
            dst, 
            g_hostBuffers[stagingIndex], 
            thisCopySize, 
            NULL ) );
        CUDA_CHECK( hipEventRecord( 
            g_events[dstDevice][stagingIndex], 
            0 ) );

        CUDA_CHECK( hipCtxPopCurrent( &dstContext ) );

        dst += thisCopySize;
        src += thisCopySize;
        N -= thisCopySize;
        stagingIndex = 1 - stagingIndex;
    }

    // Wait until both devices are done
    CUDA_CHECK( hipCtxPushCurrent( srcContext ) );
    CUDA_CHECK( hipCtxSynchronize() );
    CUDA_CHECK( hipCtxPopCurrent( &srcContext ) );

    CUDA_CHECK( hipCtxPushCurrent( dstContext ) );
    CUDA_CHECK( hipCtxSynchronize() );
    CUDA_CHECK( hipCtxPopCurrent( &dstContext ) );
    
Error:
    return status;
}

bool
TestMemcpy( 
    int *dst, int dstDevice,
    int *src, int srcDevice,
    int *srcHost, const int *srcOriginal,
    size_t dstOffset, size_t srcOffset, 
    size_t numInts )
{
    hipError_t status;
    hipCtx_t srcContext, dstContext;

    memset( srcHost, 0, numInts );
    hipSetDevice( dstDevice );
    if ( hipSuccess != hipCtxGetCurrent( &dstContext ) )
        return false;
    hipSetDevice( srcDevice );
    if ( hipSuccess != hipCtxGetCurrent( &srcContext ) )
        return false;

    CUDART_CHECK( hipMemcpy( src+srcOffset, srcOriginal+srcOffset, 
        numInts*sizeof(int), hipMemcpyHostToDevice ) );
    memset( srcHost, 0, numInts*sizeof(int) );
    chMemcpyPeerToPeer( dst+dstOffset, dstContext, dstDevice, 
                        src+srcOffset, srcContext, srcDevice, 
                        numInts*sizeof(int) );
    CUDART_CHECK( hipMemcpy( srcHost, dst+dstOffset, numInts*sizeof(int), hipMemcpyDeviceToHost ) );
    for ( size_t i = 0; i < numInts; i++ ) {
        if ( srcHost[i] != srcOriginal[srcOffset+i] ) {
            return false;
        }
    }
    return true;
Error:
    return false;
}

int
main( int argc, char *argv[] )
{
    int deviceCount;

    hipError_t status;
    int *deviceInt[2];
    int *hostInt = 0;
    const size_t numInts = 8*1048576;
    const int cIterations = 10;
    int *testVector = 0;

    hipCtx_t srcContext, dstContext;


    printf( "Peer-to-peer memcpy... " ); fflush( stdout );

    chTimerTimestamp start, stop;

    memset( deviceInt, 0, sizeof(deviceInt) );

    CUDART_CHECK( hipGetDeviceCount( &deviceCount ) );

    if ( deviceCount <= 1 ) {
        printf( "Peer-to-peer demo requires at least 2 devices\n" );
        exit(1);
    }

    for ( int i = 0; i < 2; i++ ) {
        hipSetDevice( i );

        CUDART_CHECK( hipEventCreate( &g_events[i][0] ) );
        CUDART_CHECK( hipEventRecord( g_events[i][0], 0 ) );  // so it is signaled on first synchronize
        CUDART_CHECK( hipEventCreate( &g_events[i][1] ) );
        CUDART_CHECK( hipEventRecord( g_events[i][1], 0 ) );  // so it is signaled on first synchronize

        CUDART_CHECK( hipMalloc( &deviceInt[i], numInts*sizeof(int) ) );
    }

    CUDART_CHECK( hipHostAlloc( &g_hostBuffers[0], STAGING_BUFFER_SIZE, hipHostMallocPortable ) );
    CUDART_CHECK( hipHostAlloc( &g_hostBuffers[1], STAGING_BUFFER_SIZE, hipHostMallocPortable ) );

    CUDART_CHECK( hipHostAlloc( &hostInt, numInts*sizeof(int), 0 ) );

    testVector = (int *) malloc( numInts*sizeof(int) );
    if ( ! testVector ) {
        printf( "malloc() failed\n" );
        return 1;
    }
    for ( size_t i = 0; i < numInts; i++ ) {
        testVector[i] = rand();
    }

    CUDART_CHECK( hipSetDevice( 0 ) );
    if ( hipSuccess != hipCtxGetCurrent( &dstContext ) )
        goto Error;
    CUDART_CHECK( hipSetDevice( 1 ) );
    if ( hipSuccess != hipCtxGetCurrent( &srcContext ) )
        goto Error;
    if ( ! TestMemcpy( deviceInt[0], 0, deviceInt[1], 1, 
                       hostInt, testVector, 0, 0, numInts ) ) {
        goto Error;
    }

    for ( int i = 0; i < cIterations; i++ ) {
        size_t dstOffset = rand() % (numInts-1);
        size_t srcOffset = rand() % (numInts-1);
        size_t intsThisIteration = 1 + rand() % (numInts-max(dstOffset,srcOffset)-1);
        if ( ! TestMemcpy( deviceInt[0], 0, deviceInt[1], 1, hostInt, testVector, dstOffset, srcOffset, intsThisIteration ) ) {
            //TestMemcpy( deviceInt, hostInt, testVector, dstOffset, srcOffset, intsThisIteration );
            goto Error;
        }
    }

    chTimerGetTime( &start );
    for ( int i = 0; i < cIterations; i++ ) {
        chMemcpyPeerToPeer( deviceInt[0], dstContext, 0, deviceInt[1], srcContext, 1, numInts*sizeof(int) ) ;
    }
    CUDART_CHECK( hipDeviceSynchronize() );
    chTimerGetTime( &stop );

    {
        double MBytes = cIterations*numInts*sizeof(int) / 1048576.0;
        double MBpers = MBytes / chTimerElapsedTime( &start, &stop );

        printf( "%.2f MB/s\n", MBpers );
    }

    hipFree( deviceInt );
    hipHostFree( hostInt );
    return 0;
Error:
    printf( "Error\n" );
    return 1;
}
