#include "hip/hip_runtime.h"
/*
 *
 * stream2Streams.cu
 *
 * Formulation of stream1Async.cu that uses streams to overlap data
 * transfers and kernel processing.
 *
 * Build with: nvcc -I ../chLib stream2Streams.cu
 *
 * Copyright (c) 2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <chError.h>
#include <chCommandLine.h>

#include <stdio.h>
#include <stdlib.h>

#include "saxpyCPU.h"

//
// saxpy global function adds x[i]*alpha to each element y[i]
// and writes the result to out[i].
//
// Due to low arithmetic density, this kernel is extremely bandwidth-bound.
//

__global__ void
saxpy( float *out, const float *x, const float *y, size_t N, float alpha )
{
    for ( size_t i = blockIdx.x*blockDim.x + threadIdx.x;
                 i < N;
                 i += blockDim.x*gridDim.x ) {
        out[i] = alpha*x[i]+y[i];
    }
}

hipError_t
MeasureTimes( 
    float *msTotal,
    size_t N, 
    float alpha,
    int nStreams,
    int nBlocks, 
    int nThreads )
{
    hipError_t status;
    float *dptrOut = 0, *hptrOut = 0;
    float *dptrY = 0, *hptrY = 0;
    float *dptrX = 0, *hptrX = 0;
    hipStream_t *streams = 0;
    hipEvent_t evStart = 0;
    hipEvent_t evStop = 0;
    size_t streamStep = N / nStreams;

    if ( N % nStreams ) {
        printf( "Stream count must be evenly divisible into N\n" );
        status = hipErrorInvalidValue;
        goto Error;
    }

    streams = new hipStream_t[nStreams];
    if ( ! streams ) {
        status = hipErrorOutOfMemory;
        goto Error;
    }
    memset( streams, 0, nStreams*sizeof(hipStream_t) );
    for ( int i = 0; i < nStreams; i++ ) {
        CUDART_CHECK( hipStreamCreate( &streams[i] ) );
    }
    CUDART_CHECK( hipHostAlloc( &hptrOut, N*sizeof(float), 0 ) );
    memset( hptrOut, 0, N*sizeof(float) );
    CUDART_CHECK( hipHostAlloc( &hptrY, N*sizeof(float), 0 ) );
    CUDART_CHECK( hipHostAlloc( &hptrX, N*sizeof(float), 0 ) );

    CUDART_CHECK( hipMalloc( &dptrOut, N*sizeof(float) ) );
    CUDART_CHECK( hipMemset( dptrOut, 0, N*sizeof(float) ) );

    CUDART_CHECK( hipMalloc( &dptrY, N*sizeof(float) ) );
    CUDART_CHECK( hipMemset( dptrY, 0, N*sizeof(float) ) );

    CUDART_CHECK( hipMalloc( &dptrX, N*sizeof(float) ) );
    CUDART_CHECK( hipMemset( dptrY, 0, N*sizeof(float) ) );

    CUDART_CHECK( hipEventCreate( &evStart ) );
    CUDART_CHECK( hipEventCreate( &evStop ) );
    for ( size_t i = 0; i < N; i++ ) {
        hptrX[i] = (float) rand() / RAND_MAX;
        hptrY[i] = (float) rand() / RAND_MAX;
    }
    CUDART_CHECK( hipEventRecord( evStart, 0 ) );

    for ( int iStream = 0; iStream < nStreams; iStream++ ) {
        CUDART_CHECK( hipMemcpyAsync( dptrX+iStream*streamStep, hptrX+iStream*streamStep, streamStep*sizeof(float), hipMemcpyHostToDevice, streams[iStream] ) );
        CUDART_CHECK( hipMemcpyAsync( dptrY+iStream*streamStep, hptrY+iStream*streamStep, streamStep*sizeof(float), hipMemcpyHostToDevice, streams[iStream] ) );
    }

    for ( int iStream = 0; iStream < nStreams; iStream++ ) {
        saxpy<<<nBlocks, nThreads, 0, streams[iStream]>>>( dptrOut+iStream*streamStep, dptrX+iStream*streamStep, dptrY+iStream*streamStep, streamStep, alpha );
    }

    for ( int iStream = 0; iStream < nStreams; iStream++ ) {
        CUDART_CHECK( hipMemcpyAsync( hptrOut+iStream*streamStep, dptrOut+iStream*streamStep, streamStep*sizeof(float), hipMemcpyDeviceToHost, streams[iStream] ) );
    }

    CUDART_CHECK( hipEventRecord( evStop, 0 ) );
    CUDART_CHECK( hipDeviceSynchronize() );
    for ( size_t i = 0; i < N; i++ ) {
        if ( fabsf( hptrOut[i] - (alpha*hptrX[i]+hptrY[i]) ) > 1e-5f ) {
            status = hipErrorUnknown;
            goto Error;
        }
    }
    CUDART_CHECK( hipEventElapsedTime( msTotal, evStart, evStop ) );
Error:
    if ( streams ) {
        for ( int i = 0; i < nStreams; i++ ) {
            hipStreamDestroy( streams[i] );
        }
        delete[] streams;
    }
    hipEventDestroy( evStart );
    hipEventDestroy( evStop );
    hipFree( dptrX );
    hipFree( dptrY );
    hipHostFree( hptrX );
    hipHostFree( hptrY );
    return status;
}

double
Bandwidth( float ms, double NumBytes )
{
    return NumBytes / (1000.0*ms);
}

int
main( int argc, char *argv[] )
{
    hipError_t status;
    int N_Mfloats = 128;
    size_t N;
    int nStreams = 8;
    int nBlocks = 1500;
    int nThreads = 256;
    float alpha = 2.0f;

    chCommandLineGet( &nBlocks, "nBlocks", argc, argv );
    chCommandLineGet( &nThreads, "nThreads", argc, argv );
    chCommandLineGet( &nThreads, "nStreams", argc, argv );
    chCommandLineGet( &N_Mfloats, "N", argc, argv );
    printf( "Measuring times with %dM floats", N_Mfloats );
    if ( N_Mfloats==128 ) {
        printf( " (use --N to specify number of Mfloats)");
    }
    printf( "\n" );

    N = 1048576*N_Mfloats;

    CUDART_CHECK( hipSetDeviceFlags( hipDeviceMapHost ) );
    {
        float msTotal;
        CUDART_CHECK( MeasureTimes( &msTotal, N, alpha, nStreams, nBlocks, nThreads ) );
        printf( "Total time: %.2f ms (%.2f MB/s)\n", msTotal, Bandwidth( msTotal, 3*N*sizeof(float) ) );
    }

Error:
    if ( status == hipErrorOutOfMemory ) {
        printf( "Memory allocation failed\n" );
    }
    else if ( hipSuccess != status ) {
        printf( "Failed\n" );
    }
    return hipSuccess != status;
}
