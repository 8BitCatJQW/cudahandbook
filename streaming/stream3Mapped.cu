#include "hip/hip_runtime.h"
/*
 *
 * stream3Mapped.cu
 *
 * Formulation of stream1Async.cu that uses mapped pinned memory to
 * hold the input and output data.  Since the kernel can use mapped
 * pinned memory to initiate DMA transfers across the bus, this
 * version is simpler (no hipMemcpy() calls) and just as fast.
 *
 * Build with: nvcc -I ../chLib stream3Mapped.cu
 *
 * Copyright (c) 2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <chError.h>
#include <chCommandLine.h>

#include <stdio.h>
#include <stdlib.h>

//
// saxpy global function adds x[i]*alpha to each element y[i]
// and writes the result to out[i].
//
// Due to low arithmetic density, this kernel is extremely bandwidth-bound.
//

__global__ void
saxpy( float *out, const float *x, const float *y, size_t N, float alpha )
{
    for ( size_t i = blockIdx.x*blockDim.x + threadIdx.x;
                 i < N;
                 i += blockDim.x*gridDim.x ) {
        out[i] += alpha*x[i]+y[i];
    }
}

void
saxpyCPU( float *out, const float *x, const float *y, size_t N, float alpha )
{
    for ( size_t i = 0; i < N; i++ ) {
        out[i] += alpha*x[i]+y[i];
    }
}

hipError_t
MeasureTimes( 
    float *msTotal,
    size_t N, 
    float alpha,
    int nBlocks, 
    int nThreads )
{
    hipError_t status;
    float *dptrOut = 0, *hptrOut = 0;
    float *dptrY = 0, *hptrY = 0;
    float *dptrX = 0, *hptrX = 0;
    hipEvent_t evStart = 0;
    hipEvent_t evStop = 0;

    CUDART_CHECK( hipHostAlloc( &hptrOut, N*sizeof(float), hipHostMallocMapped ) );
    CUDART_CHECK( hipHostGetDevicePointer( &dptrOut, hptrOut, 0 ) );
    memset( hptrOut, 0, N*sizeof(float) );
    CUDART_CHECK( hipHostAlloc( &hptrY, N*sizeof(float), hipHostMallocMapped ) );
    CUDART_CHECK( hipHostGetDevicePointer( &dptrY, hptrY, 0 ) );
    CUDART_CHECK( hipHostAlloc( &hptrX, N*sizeof(float), hipHostMallocMapped ) );
    CUDART_CHECK( hipHostGetDevicePointer( &dptrX, hptrX, 0 ) );

    CUDART_CHECK( hipEventCreate( &evStart ) );
    CUDART_CHECK( hipEventCreate( &evStop ) );
    for ( size_t i = 0; i < N; i++ ) {
        hptrX[i] = (float) rand() / RAND_MAX;
        hptrY[i] = (float) rand() / RAND_MAX;
    }
    CUDART_CHECK( hipEventRecord( evStart, 0 ) );
        saxpy<<<nBlocks, nThreads>>>( dptrOut, dptrX, dptrY, N, alpha );
    CUDART_CHECK( hipEventRecord( evStop, 0 ) );
    CUDART_CHECK( hipDeviceSynchronize() );
    for ( size_t i = 0; i < N; i++ ) {
        if ( fabsf( hptrOut[i] - (alpha*hptrX[i]+hptrY[i]) ) > 1e-5f ) {
            status = hipErrorUnknown;
            goto Error;
        }
    }
    CUDART_CHECK( hipEventElapsedTime( msTotal, evStart, evStop ) );
Error:
    hipEventDestroy( evStop );
    hipEventDestroy( evStart );
    hipHostFree( hptrOut );
    hipHostFree( hptrX );
    hipHostFree( hptrY );
    return status;
}

double
Bandwidth( float ms, double NumBytes )
{
    return NumBytes / (1000.0*ms);
}

int
main( int argc, char *argv[] )
{
    hipError_t status;
    int N_Mfloats = 128;
    size_t N;
    int nBlocks = 1500;
    int nThreads = 256;
    float alpha = 2.0f;

    chCommandLineGet( &nBlocks, "nBlocks", argc, argv );
    chCommandLineGet( &nThreads, "nThreads", argc, argv );
    chCommandLineGet( &N_Mfloats, "N", argc, argv );
    printf( "Measuring times with %dM floats", N_Mfloats );
    if ( N_Mfloats==128 ) {
        printf( " (use --N to specify number of Mfloats)");
    }
    printf( "\n" );

    N = 1048576*N_Mfloats;

    CUDART_CHECK( hipSetDeviceFlags( hipDeviceMapHost ) );
    {
        float msTotal;
        CUDART_CHECK( MeasureTimes( &msTotal, N, alpha, nBlocks, nThreads ) );
        printf( "Total time: %.2f ms (%.2f MB/s)\n", msTotal, Bandwidth( msTotal, 3*N*sizeof(float) ) );
    }

Error:
    if ( status == hipErrorOutOfMemory ) {
        printf( "Memory allocation failed\n" );
    }
    else if ( hipSuccess != status ) {
        printf( "Failed\n" );
    }
    return hipSuccess != status;
}
